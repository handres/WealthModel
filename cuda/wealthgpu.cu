#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*                                                                            */
/*----------------------------------------------------------------------------*/
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <string>
using namespace std;
using namespace thrust::placeholders;

/*----------------------------------------------------------------------------*/
#define EPS 1e-5
#define ISSMALL(_e) (-EPS<=(_e) && (_e)<= EPS)
#define EQUAL(_a,_b) (((_a)+(_b)==0)?true:ISSMALL(((_a)-(_b))/((_a)+(_b))))
/*----------------------------------------------------------------------------*/
typedef float WTYPE;
#define WTSZ sizeof(WTYPE)

/*----------------------------------------------------------------------------*/
struct MyTimer { hipEvent_t start, stop; float dt; };
#define TIMER_TYPE MyTimer
#define TIMER_START(_x) do{hipEventCreate(&(_x).start);hipEventCreate(&(_x).stop);hipEventRecord((_x).start,0);}while(0)
#define TIMER_END(_x) do{hipEventRecord((_x).stop,0);hipEventSynchronize((_x).stop); hipEventElapsedTime(&((_x).dt), (_x).start, (_x).stop);}while(0)
#define TIMER_SECS(_x) (((_x).dt)*1e-3)

/*----------------------------------------------------------------------------*/
static int dbglvl = 1;
#define DBG(_l,_s) if((_l)<=dbglvl){cout<<_s;}else
#define DBGNL(_l,_s) DBG(_l,_s<<endl)

/*----------------------------------------------------------------------------*/
bool normalize = false, verify = true, writeresults = true;

/*----------------------------------------------------------------------------*/
WTYPE initw( WTYPE *w, long n, WTYPE iw )
{
    WTYPE wsum = 0;
    srand48(345);
    for( long i = 0; i < n; i++ )
    {
        w[i] = (iw >= 0 ? iw : (-iw)*drand48());
        wsum += w[i];
    }
    return wsum;
}

/*----------------------------------------------------------------------------*/
void printw( int d, const char *s, WTYPE *w, long n )
{
    DBG(d,"** "<<s<<": ");
    for( long i = 0; i < n; i++ )
    {
        DBG(d,(i>0?", ":"")<<w[i]);
        if( i >= 4 && n > 10 && i < n-6 )
        {
            DBG(d,", ...");
            i = n-6;
        }
    }
    DBGNL(d,"");
}

/*----------------------------------------------------------------------------*/
void writew( WTYPE *w, long n, long ts )
{
    char fname[100];
    sprintf( fname, "wealth-npersons-%ld-at-time-%ld.bin", n, ts );
    DBGNL(0,"Writing wealth distribution to file \""<<fname<<"\"");
    ofstream fout;
    fout.open( fname, ios::binary | ios::out );
    fout.write( (char*) w, n*sizeof(WTYPE) );
    fout.close();
    DBGNL(2,"Writing done");
}

/*----------------------------------------------------------------------------*/
long nthr = 16*32, nblk = 32;
long totthr = 0;
hiprandState *randstate = 0;

/*----------------------------------------------------------------------------*/
#define randnext( _rs, _i ) hiprand_uniform( &(_rs)[(_i)] )

/*----------------------------------------------------------------------------*/
__global__
void k_enrich( WTYPE *w, WTYPE *wdf,
               long n, long npt,
               WTYPE sumbeta, WTYPE dw,
               long ndw, hiprandState *rs )
{
    long nthr = (gridDim.x * blockDim.x);
    long tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    long nr = ndw/nthr;
    if( tid < ndw%nthr ) nr++;

    for( long r = 0; r < nr; r++ )
    {
      float rng = randnext( rs, tid );
      WTYPE val = rng*sumbeta;
      long si = 0, ei = n-1;
      //long si = tid*npt, ei = si+npt-1;
      if( ei >= n )
      {
        ei = n-1;
      }
      if( si < n )
      {
        long mi = -1;
        if( val < wdf[si] )
        {
            WTYPE prev = (si > 0 ? wdf[si-1] : 0.0);
            if( prev < val )
            {
                mi = si;
            }
        }
        else if( val <= wdf[ei] )
        {
            mi = si+(ei-si)/2;
            while( si < ei )
            {
                if( wdf[mi] == val )
                {
                    break;
                }
                else if( val < wdf[mi] )
                {
                    ei = mi-1;
                }
                else
                {
                    si = mi+1;
                }
                mi = si+(ei-si)/2;
            }
        }
        if( mi >= 0 )
        {
            atomicAdd( &w[mi], dw );
        }
      }
    }
}

/*----------------------------------------------------------------------------*/
void enrich( WTYPE *w, WTYPE *wdf, long n, WTYPE sumw2, WTYPE dw, long ndw )
{
    long npt = (n <= totthr) ? 1 : (1+(n-1)/totthr);
    DBGNL(2,"npt="<<npt);
    DBGNL(1,"val="<<sumw2);
    k_enrich<<<nblk,nthr>>>( w, wdf, n, npt, sumw2, dw, ndw, randstate );
}

/*----------------------------------------------------------------------------*/
__global__
void k_initrand( hiprandState *randstate )
{
    long tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    long rseed = 1234;
    hiprand_init( rseed, tid, 0, &randstate[tid] );
}

/*----------------------------------------------------------------------------*/
void initrand( void )
{
    totthr = nblk*nthr;
    hipMalloc( &randstate, totthr*sizeof(randstate[0]) );
    k_initrand<<<nblk,nthr>>>( randstate );
}

/*----------------------------------------------------------------------------*/
class PowerFunctor
{
    WTYPE e;
    public: PowerFunctor(WTYPE _e){e=_e;}
    public: __host__ __device__ WTYPE operator()(WTYPE x)const{return pow(x,e);}
};

/*----------------------------------------------------------------------------*/
int main( int ac, char *av[] )
{
    long np = 10, niter = 10, ndw = 1024;
    float beta = 1.36;
    WTYPE iw = 1.0, dw = 0.1;
    if(getenv("VERIFY"))verify=!strcmp(getenv("VERIFY"),"TRUE");
    if(getenv("NORMALIZE"))normalize=!strcmp(getenv("NORMALIZE"),"TRUE");
    if(getenv("WRITERESULTS"))writeresults=!strcmp(getenv("WRITERESULTS"),"TRUE");
    if(getenv("DEBUG"))dbglvl=atoi(getenv("DEBUG"));
    if(getenv("BETA"))beta=atof(getenv("BETA"));
    if( ac <= 1 )
    {
        DBGNL(0,"Usage: "<<av[0]<<" npersons niterations initwealth deltawealth ndw beta");
        DBGNL(0,"All persons have same given init wealth if initwealth > 0 ");
        DBGNL(0,"else (if initwealth < 0), have random wealth with mean=-initwealth");
        exit(1);
    }
    if( ac >= 2 ) np = atoi( av[1] );
    if( ac >= 3 ) niter = atoi( av[2] );
    if( ac >= 4 ) iw = atof( av[3] );
    if( ac >= 5 ) dw = atof( av[4] );
    if( ac >= 6 ) ndw = atoi( av[5] );
    if( ac >= 7 ) beta = atof( av[6] );

    DBGNL(0,"dbglvl="<<dbglvl);
    DBGNL(0,"verify="<<verify);
    DBGNL(0,"normalize="<<normalize);
    DBGNL(0,"writeresults="<<writeresults);
    DBGNL(0,"npersons="<<np<<" niterations="<<niter<<" iw="<<iw<<" dw="<<dw<<" beta="<<beta);

    long nbytes = np*WTSZ;
    WTYPE *h_w = (WTYPE*)malloc( nbytes ); DBGNL(1,"mallocated w"<<nbytes);
    WTYPE *h_wdf = (WTYPE*)malloc( nbytes ); DBGNL(2,"mallocated wdf"<<nbytes);
    WTYPE h_totw = initw( h_w, np, iw );
    printw( 0, "Initialized", h_w, np );

    initrand(); DBGNL(0,"RNG initialized");
    WTYPE *d_w = 0, *d_wdf = 0;
    hipMalloc( &d_w, nbytes ); DBGNL(2,"w cudaMallocated "<<nbytes);
    hipMalloc( &d_wdf, nbytes ); DBGNL(3,"wdf cudaMallocated "<<nbytes);
    hipMemcpy( d_w, h_w, nbytes, hipMemcpyHostToDevice ); DBGNL(3,"H2D "<<nbytes);
    thrust::device_ptr<WTYPE> td_w(d_w), td_wdf(d_wdf);

    #define GPW(_d,_s) if((_d)<=dbglvl){ \
        hipMemcpy( h_w, d_w, nbytes, hipMemcpyDeviceToHost ); DBGNL(10,"D2H W"<<nbytes); \
        printw( _d, _s, h_w, np ); \
      }else
    #define GPWDF(_d,_s) if((_d)<=dbglvl){ \
        hipMemcpy( h_wdf, d_wdf, nbytes, hipMemcpyDeviceToHost ); DBGNL(10,"D2H WDF"<<nbytes); \
        printw( _d, _s, h_wdf, np ); \
      }else

    long nprints = getenv("NPRINTS") ? atoi(getenv("NPRINTS")) : 100;
    long printevery = niter<nprints?1:(niter/nprints);
    DBGNL(0,"nprints="<<nprints);
    long nwrites = getenv("NWRITES") ? atoi(getenv("NWRITES")) : 10;
    long writeevery = niter<nwrites?1:(niter/nwrites);
    DBGNL(0,"nwrites="<<nwrites);

    DBGNL(0,"Starting simulation...");
    TIMER_TYPE timer;
    TIMER_START(timer);
    thrust::sort( td_w, td_w+np ); GPW(3,"Sorted");
    for( long iter = 0; iter < niter; iter++ )
    {
        if((iter+1)%printevery==0)DBGNL(2,"------ Iteration "<<iter<<" Start ------");
        thrust::transform( td_w, td_w+np, td_wdf, PowerFunctor(beta) ); GPWDF(4,"Exponentiated");
        WTYPE h_totwdf = thrust::reduce( td_wdf, td_wdf+np ); DBGNL(1,"Totwdf="<<h_totwdf);
        if( normalize )
        {
            WTYPE h_totwdf_inv = ( (h_totwdf == 0) ? 0.0 : (1.0 / h_totwdf));
            thrust::transform( td_wdf, td_wdf+np, td_wdf, _1*h_totwdf_inv ); GPWDF(5,"Normalized");
            if(verify)
            {
                WTYPE hsum = thrust::reduce( td_wdf, td_wdf+np ); DBGNL(1,"Totwdfnorm="<<hsum);
                if( !EQUAL(hsum,1.0) ) { cerr<<"SUM MISMATCH "<<(hsum-1.0)<<endl; exit(1); }
            }
        }
        thrust::inclusive_scan( td_wdf, td_wdf+np, td_wdf ); GPWDF(6,"Scanned");
        enrich( d_w, d_wdf, np, h_totwdf, dw, ndw ); GPW(2,"Enriched");
        thrust::sort( td_w, td_w+np ); GPW(3,"Sorted");
        if( verify || iter==niter-1 )
        {
            WTYPE hsum = thrust::reduce( td_w, td_w+np ); DBGNL(1,"Totw="<<hsum);
            WTYPE isum = h_totw + (iter+1)*ndw*dw;
            if( !EQUAL(hsum,isum) ) { cerr<<"SUM MISMATCH "<<hsum<<" != "<<isum<<" ("<<(hsum-isum)<<")"<<endl; if(verify)exit(1); }
            DBGNL(0,"Total wealth="<<isum);
        }
        if( (iter+1)%writeevery==0 || iter==niter-1 )
        {
            if( writeresults )
            {
                GPW(0,"Writing");
                writew( h_w, np, iter+1 );
            }
        }
        if((iter+1)%printevery==0)DBGNL(0,"------ Iteration "<<iter<<" ("<<((iter+1)*100.0/niter)<<"%) End ------");
    }
    TIMER_END(timer);

    float dt = TIMER_SECS(timer);
    float million = 1e6;
    float thousand = 1e3;
    DBGNL(0,"           Iterations = "<<niter/million<<" million");
    DBGNL(0,"              Persons = "<<np/million<<" million");
    DBGNL(0," #Wealth packets/Iter = "<<ndw/thousand<<" thousand");
    DBGNL(0,"Total #wealth packets = "<<ndw*niter/million<<" million");
    DBGNL(0,"                 Time = "<<dt<<" secs");
    DBGNL(0,"       Time/Iteration = "<<(dt/niter)*1e3<<" millisecs");
    DBGNL(0,"Time/Iteration/Person = "<<(dt/niter/np)*1e6<<" microsecs");

    GPW(0,"Final enrichment");

    DBGNL(0,"End");
}

/*----------------------------------------------------------------------------*/
